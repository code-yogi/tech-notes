
#include <hip/hip_runtime.h>
#include <stdio.h>

#define NUM_BLOCKS 8
#define BLOCK_SIZE 64
#define WINDOW_SIZE 3
#define NUM_ELEMENTS (NUM_BLOCKS * BLOCK_SIZE)

__global__ void mykernel(int *xp, int *result) {
  int globalIdx = (blockIdx.x * blockDim.x) + threadIdx.x;
  int localIdx = threadIdx.x + WINDOW_SIZE;

  // Keep a local buffer that's faster to read from.
  // nvcc really wants this to be a constant sized array. Thus I had
  // to make BLOCK_SIZE a constant.
  __shared__ int buffer[BLOCK_SIZE + 2 * WINDOW_SIZE];
  buffer[localIdx] = xp[globalIdx];
  // At edge of block may have to copy extra elements. Unless at the
  // edge of the entire array.
  if ((threadIdx.x < WINDOW_SIZE) && (0 <= (globalIdx - WINDOW_SIZE))) {
    buffer[localIdx - WINDOW_SIZE] = xp[globalIdx - WINDOW_SIZE];
  }
  if (((threadIdx.x + WINDOW_SIZE) >= blockDim.x)
      && ((globalIdx + WINDOW_SIZE) < NUM_ELEMENTS)) {
    buffer[localIdx + WINDOW_SIZE] = xp[globalIdx + WINDOW_SIZE];
  }
  // Need this so everyone pauses and doesn't race ahead.
  __syncthreads();

  int resultValue = 0;
  for (int i = -WINDOW_SIZE; i <= WINDOW_SIZE; i++) {
    resultValue += buffer[localIdx + i];
  }

  result[globalIdx] = resultValue;

  return;
}

int expectedResult(int *xp, int position) {
  int expectedResult_ = 0;

  for (int i = -WINDOW_SIZE; i <= WINDOW_SIZE; i++) {
    if ((position + i) < 0) continue;
    if ((position + i) >= NUM_ELEMENTS) continue;
    expectedResult_ += xp[position + i];
  }

  return expectedResult_;
}

int main() {
  int arrayBytes = NUM_ELEMENTS * sizeof(int);

  // Allocate and set host memory.
  int *xp = (int*) malloc(arrayBytes);
  int *result = (int*) malloc(arrayBytes);
  for (int i = 0; i < NUM_ELEMENTS; i++) {
    xp[i] = i * i;
  }

  // Allocate and set device memory.
  int *deviceXp;
  int *deviceResult;
  hipMalloc(&deviceXp, arrayBytes);
  hipMalloc(&deviceResult, arrayBytes);
  hipMemcpy(deviceXp, xp, arrayBytes, hipMemcpyHostToDevice);

  // Run kernel and copy result back to host.
  mykernel<<<NUM_BLOCKS, BLOCK_SIZE>>>(deviceXp, deviceResult);
  hipMemcpy(result, deviceResult, arrayBytes, hipMemcpyDeviceToHost);

  // Verify result.
  for (int i = 0; i < NUM_ELEMENTS; i++) {
    int expectedResult_ = expectedResult(xp, i);
    if (result[i] == expectedResult_) continue;
    printf("result[%d] = %d != %d\n", i, result[i], expectedResult_);
  }

  // Free all the memory!
  free(xp);
  hipFree(deviceXp);
  free(result);
  hipFree(deviceResult);

  return 0;
}
