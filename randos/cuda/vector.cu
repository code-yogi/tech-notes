
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void mykernel(int* a, int* b, int* c) {
  int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
  c[idx] = a[idx] * b[idx];
}

// Probably needs to be a define since we'll use it in <<<
#define NUM_BLOCKS 8
#define NUM_THREADS_PER_BLOCK 64

int main() {
  // Host
  int *a, *b, *c;
  // Device pointers
  int *d_a, *d_b, *d_c;
  int numElements = NUM_BLOCKS * NUM_THREADS_PER_BLOCK;
  int arraySize = numElements * sizeof(int);

  // Allocate and initialize host memory.
  a = (int *) malloc(arraySize);
  b = (int *) malloc(arraySize);
  c = (int *) malloc(arraySize);
  for (int i = 0; i < numElements; i++) {
    a[i] = i;
    b[i] = numElements - i;
  }

  // Allocate memory on device. Store pointers on host.
  // Pretty sure I don't need void** cast here.
  hipMalloc((void**) &d_a, arraySize);
  hipMalloc((void**) &d_b, arraySize);
  hipMalloc((void**) &d_c, arraySize);

  // Copy host values to device.
  hipMemcpy(d_a, a, arraySize, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, arraySize, hipMemcpyHostToDevice);

  // Run kernel.
  mykernel<<<NUM_BLOCKS, NUM_THREADS_PER_BLOCK>>>(d_a, d_b, d_c);

  // Copy device values to host
  hipMemcpy(c, d_c, arraySize, hipMemcpyDeviceToHost);

  // Deallocate memory.
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  for (int i = 0; i < numElements; i++) {
    printf("Result %d: %d = %d * %d\n", i, c[i], a[i], b[i]);
  }

  return 0;
}
